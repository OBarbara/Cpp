/*
 ============================================================================
 Name        : NTTFFTCUDA.cu
 Author      : Owen
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <stdint.h>
#include <math.h>

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
void ProcessandTime(const int size, const int bpe, const int thread_limit);
void initialize(uint32_t* arr, int size, const int bpe);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)
/**
 * CUDA kernel that computes the integer FFT of an array of unsigned integers
 */
// both of the following constants must be a power of two
// the first part takes care of the first stages before the shuffling and the
// element transfer from shared memory to global memory
__global__ void MulSMNTTKernel1(uint32_t *data, const short gridSizeSh, const short th_sec, short curr_k, short k, int mod, const uint32_t* g, bool repos) {
	unsigned short idx = threadIdx.x;
	unsigned short b = blockIdx.x;
	const int BLOCK_SIZE = blockDim.x;	// the offset given for the second part of shared memory
	if (idx < BLOCK_SIZE){	// for maximum performance, the last idx should be a multiple of 32 (minus 1 since it starts from 0)
		uint32_t tempvar; 	  // a temporary variable to manage subtractions and hold temporary data
		short i = idx >> 1;
		short j = idx & 1;	// even or odd thread
		// used for twiddle factor indexing
		short l = k - curr_k - 1;
		short m = i & 1;	// helps in indexing for each thread in a block
		short elem;
		short b_twid = b << (l - 1);	// twiddle factor indexing for each block
		if(BLOCK_SIZE >= gridDim.x)
			elem = idx + (b << th_sec) + (((idx >> th_sec) * (gridDim.x - 1)) << th_sec);
		else
			elem = b + (idx << gridSizeSh);
		short p, tmp, x, offx, x1, twid_1, twid_2, twid_index = 0;
		// sign takes care of the operations for even (addition)
		// and odd (subtraction) threads
		short sign = ((-2) * j) + 1;  // can be 1 or -1
		// temporary array to store intermediate data
		extern __shared__ uint32_t final_temp[];
		extern __shared__ uint32_t temp_data[];

		// copy data from global memory to shared memory
		final_temp[idx] = data[elem];
		__syncthreads();	// ensure that the shared memory is properly populated


		// stg is the stage (or epoch)
		// k is the number of stages in the computation
		for(short stg = 1; stg <= curr_k; stg++){
			p = BLOCK_SIZE >> stg;	// this variable will help in indexing
			// indexing
			tmp = i + ((i >> (k - curr_k - stg)) * (1 << (k - curr_k - stg)));
			x = tmp + (j * p);
			offx = BLOCK_SIZE + x;	// use this to index second part of shared memory (temp_data[])
			x1 = x + (sign * p);
			if(repos){
				twid_1 = (b + (i << gridSizeSh)) * (curr_k - stg);		// twiddle for stage 1
				twid_2 = ((l - 1) * ((!m * b_twid) + (m * (b_twid + BLOCK_SIZE))) + (curr_k - l) * (b << l)) * (stg - 1);	// twiddle for stage 2
				// twid_index manages the indexing of the twiddle factors
				twid_index =  twid_1 + twid_2;
			}
			// since the value should be unsigned, a subtraction cannot result in a negative number
			// so we add the modulus to the number being subtracted to prevent that from happening
			tempvar = final_temp[x1] + mod;
			// addition and subtraction is taken care of here
			// modulus is done after addition/subtraction
			temp_data[offx] = (tempvar + (sign * final_temp[x])) % mod;
			// shift by twiddle factor and perform modulus
			if(repos){
				temp_data[offx] <<= j * g[twid_index];
				temp_data[offx] %= mod;
			}
			final_temp[x] = temp_data[offx];
			__syncthreads();
		}
		// shuffle data from shared to global memory
		data[elem] = final_temp[idx];
	}
}

// the first part takes care of the first stages before the shuffling and the
// element transfer from shared memory to global memory
__global__ void MulSMNTTKernel2(uint32_t *data, int NTT_SIZE, const short gridSizeSh, short curr_k, short k, int mod, const uint32_t* g, bool repos) {
	unsigned short idx = threadIdx.x;
	unsigned short b = blockIdx.x;
	unsigned short glob_idx = b * blockDim.x + idx;
	const int BLOCK_SIZE = blockDim.x;	// the offset given for the second part of shared memory
	if (idx < BLOCK_SIZE){	// for maximum performance, the last idx should be a multiple of 32 (minus 1 since it starts from 0)
		uint32_t tempvar; 	  // a temporary variable to manage subtractions and hold temporary data
		short i = idx >> 1;
		short j = idx & 1;	// even or odd thread
		short p, tmp, x, offx, x1, twid_index;
		// sign takes care of the operations for even (addition)
		// and odd (subtraction) threads
		short sign = ((-2) * j) + 1;  // can be 1 or -1
		// temporary array to store intermediate data
		extern __shared__ uint32_t final_temp[];
		extern __shared__ uint32_t temp_data[];

		// copy data from global memory to shared memory
		final_temp[idx] = data[glob_idx];
		__syncthreads();	// ensure that the shared memory is properly populated
		// stg is the stage (or epoch)
		// k is the number of stages in the computation
		for(short stg = curr_k; stg <= k; stg++){
			p = NTT_SIZE >> stg;	// this variable will help in indexing
			// indexing
			tmp = i + ((i >> (k - stg)) * (1 << (k - stg)));
			x = tmp + (j * p);
			offx = BLOCK_SIZE + x;
			x1 = x + (sign * p);
			// twid_index manages the indexing of the twiddle factors
			twid_index = (i % p) << (stg - 1);
			// since the value should be unsigned, a subtraction cannot result in a negative number
			// so we add the modulus to the number being subtracted to prevent that from happening
			tempvar = final_temp[x1] + mod;

			// addition and subtraction is taken care of here
			// modulus is done after addition/subtraction
			temp_data[offx] = (tempvar + (sign * final_temp[x])) % mod;
			// shift by twiddle factor and perform modulus
			if(repos){
				temp_data[offx] <<= j * g[twid_index];
				temp_data[offx] %= mod;
			}
			final_temp[x] = temp_data[offx];
			__syncthreads();
		}
		// write finished data from shared to global memory
		data[glob_idx] = final_temp[idx];
	}
}

// kernel for the inverse NTT FFT
__global__ void MulSMINTTKernel1(uint32_t *data, const int NTT_SIZE, const short BITS_PER_ELEM, const short gridSizeSh, short curr_k, short k, int mod, const uint32_t* g, bool repos, bool fin) {
	unsigned short idx = threadIdx.x;
	unsigned short b = blockIdx.x;
	unsigned short glob_idx = b * blockDim.x + idx;
	const int BLOCK_SIZE = NTT_SIZE >> gridSizeSh;	// the offset given for the second part of shared memory
	if (idx < BLOCK_SIZE){	// for maximum performance, the last idx should be a multiple of 32 (minus 1 since it starts from 0)
		uint32_t tempvar; 	  // a temporary variable to manage subtractions and hold temporary data
		short i = idx >> 1;
		short j = idx & 1;	// even or odd thread
		short p, tmp, x, offx, x1, twid_index;
		// sign takes care of the operations for even (addition)
		// and odd (subtraction) threads
		short sign = ((-2) * j) + 1;  // can be 1 or -1
		// temporary array to store intermediate data
		extern __shared__ uint32_t final_temp[];
		extern __shared__ uint32_t temp_data[];

		// copy data from global memory to shared memory
		final_temp[idx] = data[glob_idx];
		__syncthreads();	// ensure that the shared memory is properly populated
		// stg is the stage (or epoch)
		// k is the number of stages in the computation
		for(short stg = k; stg >= curr_k; stg--){
			p = NTT_SIZE >> stg;	// this variable will help in indexing
			// indexing
			tmp = i + ((i >> (k - stg)) * (1 << (k - stg)));
			x = tmp + (j * p);
			offx = BLOCK_SIZE + x;
			x1 = x + (sign * p);
			// twid_index manages the indexing of the twiddle factors
			if(repos){
				twid_index = (i % p) << (stg - 1);
				// shift by twiddle factor and perform modulus
				tempvar = (final_temp[x1] << (!j * g[twid_index])) % mod;
				temp_data[offx] = final_temp[x] << (j * g[twid_index]);
				temp_data[offx] %= mod;
			}
			else{
				tempvar = final_temp[x1];
				temp_data[offx] = final_temp[x];
			}

			// since the value should be unsigned, a subtraction cannot result in a negative number
			// so we add the modulus to the number being subtracted to prevent that from happening
			tempvar += mod;
			// addition and subtraction is taken care of here
			// modulus is done after addition/subtraction
			temp_data[offx] = (tempvar + (sign * temp_data[offx])) % mod;
			final_temp[x] = temp_data[offx];
			// new data is ready for next stage
			__syncthreads();
		}

		if(fin){	// if there is only 1 SM
			// divide each element by N
			short t = (BITS_PER_ELEM << 1) - k; // convert division into multiplication
			uint32_t ls, rs;
			short size = sizeof(uint32_t) << 3; // multiply 4 bytes by 8 in this case (32 bits)
			bool rt_shift = false;	// should the shift be to the right and rotate bits?
			if(t < 0){	// if t is negative, all shifts are to the right
				uint32_t mask = t >> (size - 1);     // make a mask of the sign bit
				t ^= mask;                   // toggle the bits
				t += mask & 1;               // add one
				rt_shift = true;
			}
			short temp_sh;
			short shift = size - (BITS_PER_ELEM + 1);
			// shifting will be done in an optimized manner
			if (shift > t)
				shift = t;	// if t is less than the maximum shift amount, then assign the shift amount to be t
			temp_sh = shift;
			for (short m = 0; m < t; m += shift) {
				shift = temp_sh;	// assign the shift value from the previous iteration

				// there is no thread divergence here since all threads execute the same branch
				if(!rt_shift){	// normal left shift
					final_temp[idx] <<= shift;
				}
				else{	// right shift and bit rotation
					rs = final_temp[idx] >> shift;
					ls = final_temp[idx] << (size - shift);
					final_temp[idx] = rs | ls;
				}
				final_temp[idx] %= mod;
				if (t - shift <= (BITS_PER_ELEM - 1))
					temp_sh = t - shift;	// all remaining shifts will be done after modulus
			}

			// copy final data from shared memory to global memory
			if(repos)
				data[(NTT_SIZE - glob_idx) % NTT_SIZE] = final_temp[idx];
			else
				data[glob_idx] = final_temp[idx];
		}
		else{
			// copy from shared memory to global memory in a coalesced manner
			data[glob_idx] = final_temp[idx];
		}
	}
}

__global__ void MulSMINTTKernel2(uint32_t *data, const int NTT_SIZE, const short BITS_PER_ELEM, const short gridSizeSh, const short th_sec, short curr_k, short k, int mod, const uint32_t* g, bool repos) {
	unsigned short idx = threadIdx.x;
	unsigned short b = blockIdx.x;
	const int BLOCK_SIZE = NTT_SIZE >> gridSizeSh;	// the offset given for the second part of shared memory
	if (idx < BLOCK_SIZE){	// for maximum performance, the last idx should be a multiple of 32 (minus 1 since it starts from 0)
		uint32_t tempvar; 	  // a temporary variable to manage subtractions and hold temporary data
		short i = idx >> 1;
		short j = idx & 1;	// even or odd thread
		// the following three variables are used in the computation of the twiddle factor index
		short l = k - curr_k - 1;
		short m = i & 1;
		short elem;
		short b_twid = b << (l - 1);	// twiddle factor indexing for each block

		if(BLOCK_SIZE >= gridDim.x)
			elem = idx + (b << th_sec) + (((idx >> th_sec) * (gridDim.x - 1)) << th_sec);
		else
			elem = b + (idx << gridSizeSh);
		short x, tmp, p, offx, x1, twid_1, twid_2, twid_index;
		// sign takes care of the operations for even (addition)
		// and odd (subtraction) threads
		short sign = ((-2) * j) + 1;  // can be 1 or -1
		// temporary array to store intermediate data
		extern __shared__ uint32_t final_temp[];
		extern __shared__ uint32_t temp_data[];

		// copy data from global memory to shared memory
		final_temp[idx] = data[elem];
		__syncthreads();	// ensure that the shared memory is properly populated
		// stg is the stage (or epoch)
		// k is the number of stages in the computation
		for(short stg = curr_k; stg >= 1; stg--){
			p = BLOCK_SIZE >> stg;	// this variable will help in indexing
			// indexing
			tmp = i + ((i >> (k - curr_k - stg)) * (1 << (k - curr_k - stg)));
			x = tmp + (j * p);
			offx = BLOCK_SIZE + x;
			x1 = x + (sign * p);
			if(repos){
				twid_1 = (b + (i << gridSizeSh)) * (curr_k - stg);		// twiddle for stage 1
				twid_2 = ((l - 1) * ((!m * b_twid) + (m * (b_twid + BLOCK_SIZE))) + (curr_k - l) * (b << l)) * (stg - 1);	// twiddle for stage 2

				// twid_index manages the indexing of the twiddle factors
				twid_index =  twid_1 + twid_2;
				// shift by twiddle factor and perform modulus
				tempvar = (final_temp[x1] << (!j * g[twid_index])) % mod;
				temp_data[offx] = final_temp[x] << (j * g[twid_index]);
				temp_data[offx] %= mod;
			}
			else{
				tempvar = final_temp[x1];
				temp_data[offx] = final_temp[x];
			}

			// since the value should be unsigned, a subtraction cannot result in a negative number
			// so we add the modulus to the number being subtracted to prevent that from happening
			tempvar += mod;
			// addition and subtraction is taken care of here
			// modulus is done after addition/subtraction
			temp_data[offx] = (tempvar + (sign * temp_data[offx])) % mod;
			final_temp[x] = temp_data[offx];
			// new data is ready for next stage
			__syncthreads();
		}

		// divide each element by N
		short t = (BITS_PER_ELEM << 1) - k; // convert division into multiplication
		uint32_t ls, rs;
		short size = sizeof(uint32_t) << 3; // multiply 4 bytes by 8 in this case (32 bits)
		bool rt_shift = false;	// should the shift be to the right and rotate bits?
		if(t < 0){	// if t is negative, all shifts are to the right
			uint32_t mask = t >> (size - 1);     // make a mask of the sign bit
			t ^= mask;                   // toggle the bits
			t += mask & 1;               // add one
			rt_shift = true;
		}
		short temp_sh;
		short shift = size - (BITS_PER_ELEM + 1);
		// shifting will be done in an optimized manner
		if (shift > t)
			shift = t;	// if t is less than the maximum shift amount, then assign the shift amount to be t
		temp_sh = shift;
		for (m = 0; m < t; m += shift) {
			shift = temp_sh;	// assign the shift value from the previous iteration

			// there is no thread divergence here since all threads execute the same branch
			if(!rt_shift){	// normal left shift
				final_temp[idx] <<= shift;
			}
			else{	// right shift and bit rotation
				rs = final_temp[idx] >> shift;
				ls = final_temp[idx] << (size - shift);
				final_temp[idx] = rs | ls;
			}
			final_temp[idx] %= mod;
			if (t - shift <= (BITS_PER_ELEM - 1))
				temp_sh = t - shift;	// all remaining shifts will be done after modulus
		}

		// copy final data from shared memory to global memory
		if(repos)
			data[(NTT_SIZE - elem) % NTT_SIZE] = final_temp[idx];
		else
			data[elem] = final_temp[idx];
	}
}


/*! \name GPU function caller and process timing function */
void ProcessandTime(const int size, const int bpe, const int thlimit){
	// define NTT variables
	double intervalNTT = 0, intervalINTT = 0;
	float NTTms, iNTTms;
	hipEvent_t NTTstart, NTTstop, INTTstart, INTTstop;
	hipEventCreate(&NTTstart);
	hipEventCreate(&NTTstop);
	hipEventCreate(&INTTstart);
	hipEventCreate(&INTTstop);
	short kt = log2((double)size);;
	int i;	// index for traversing through the arrays
	int modulus = (1 << bpe) + 1; // prime number for modulo arithmetic
	int rt_unity = 2 * bpe / size; // calculate # bits for root of unity
	// these flags determine whether the IFFT repositions the elements
	// and whether the result of the IFFT matches the input array or not, respectively
	bool repos_flag = true, flag = true;
	int runs = 1E3;	// number of times each kernel function is executed
	uint32_t *in = new uint32_t[size];
	uint32_t *NTT_out = new uint32_t[size];
	uint32_t *INTT_out = new uint32_t[size];
	uint32_t* gpuNTTData, *gpuINTTData, *gpuTwid;
	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuNTTData, sizeof(uint32_t) * size));
	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuTwid, sizeof(uint32_t)*size / 2));
	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuINTTData, sizeof(uint32_t)*size));
	uint32_t twiddle[size / 2];
	if(!rt_unity)
		repos_flag = false;

	for(int i = 0; i < size / 2; i++){	// pre-compute twiddle factor array
		twiddle[i] = rt_unity * i;
	}
	int blk_size;
	bool isSingleSM = false;
	if(size <= thlimit){
		blk_size = size;
		isSingleSM = true;
	}
	else
		blk_size = thlimit;
	static const int BLOCK_SIZE = blk_size;	// amount of threads in each block
	const int blockCount = (size) / BLOCK_SIZE;	// amount of blocks in a grid
	const short gridSizeSh = logf(blockCount) / logf(2);
	short coal_segment = BLOCK_SIZE >> gridSizeSh;
	coal_segment = log2((float)coal_segment);
	short k = log2((double)blockCount);	// first kernel covers first k stages only out of kt for FFT, and vice-versa for IFFT kernels
	std::cout << "Launching kernels with " << blockCount << " block(s), each with " << BLOCK_SIZE << " threads." << std::endl;
	srand(time(NULL));	// generate the seed for the pseudo-random number generator
	for (int j = 0; j < runs; j++) {
		initialize(in, modulus, size);

		CUDA_CHECK_RETURN(hipMemcpy(gpuNTTData, in, sizeof(uint32_t)*size, hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMemcpy(gpuTwid, twiddle, sizeof(uint32_t)*size/2, hipMemcpyHostToDevice));

		hipEventRecord(NTTstart);
		if(!isSingleSM){
			// computes first k stages and shuffles data from shared memory to global memory

			MulSMNTTKernel1<<<blockCount, BLOCK_SIZE, 2 * BLOCK_SIZE * sizeof(uint32_t)>>> (gpuNTTData, gridSizeSh, coal_segment, k, kt, modulus, gpuTwid, repos_flag);
		}
		// shuffles data from global memory to shared memory and computes last set of stages
		MulSMNTTKernel2<<<blockCount, BLOCK_SIZE, 2 * BLOCK_SIZE * sizeof(uint32_t)>>> (gpuNTTData, size, gridSizeSh, k + 1, kt, modulus, gpuTwid, repos_flag);
		hipEventRecord(NTTstop);

		CUDA_CHECK_RETURN(hipMemcpy(NTT_out, gpuNTTData, sizeof(uint32_t) * size, hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpy(gpuINTTData, NTT_out, sizeof(uint32_t)*size, hipMemcpyHostToDevice));

		hipEventSynchronize(NTTstop);
		hipEventElapsedTime(&NTTms, NTTstart, NTTstop);
		hipEventRecord(INTTstart);
		// computes first stages and shuffles data from shared memory to global memory
		MulSMINTTKernel1<<<blockCount, BLOCK_SIZE, 2 * BLOCK_SIZE * sizeof(uint32_t)>>> (gpuINTTData, size, bpe, gridSizeSh, k + 1, kt, modulus, gpuTwid, repos_flag, isSingleSM);

		if(!isSingleSM){
			// shuffles data from global memory to shared memory and computes last set of stages
			MulSMINTTKernel2<<<blockCount, BLOCK_SIZE, 2 * BLOCK_SIZE * sizeof(uint32_t)>>> (gpuINTTData, size, bpe, gridSizeSh, coal_segment, k, kt, modulus, gpuTwid, repos_flag);
		}
		hipEventRecord(INTTstop);

		CUDA_CHECK_RETURN(hipMemcpy(INTT_out, gpuINTTData, sizeof(uint32_t)*size, hipMemcpyDeviceToHost));

		hipEventSynchronize(INTTstop);
		hipEventElapsedTime(&iNTTms, INTTstart, INTTstop);

		// total time taken is summed up from all the runs
		intervalNTT += NTTms;
		intervalINTT += iNTTms;


		for (i = 0; i < size; i++) {
			if (in[i] != INTT_out[i]) {
				flag = false;
				break;
			}
		}

		if (!flag) {
			std::cout << "j = " << j << ", i = " << i << "\nin = " << in[i] << ", out = " << INTT_out[i] << std::endl;
			break;
		}
	}

	if (flag){
		std::cout << "NTT matched. Average time taken for NTT is: " << intervalNTT << " microseconds" << std::endl;
		std::cout << "Average time taken for INTT is: " << intervalINTT << " microseconds" << std::endl;
	}

	// clean up
	hipEventDestroy(NTTstart);
	hipEventDestroy(NTTstop);
	hipEventDestroy(INTTstart);
	hipEventDestroy(INTTstop);
	CUDA_CHECK_RETURN(hipFree(gpuNTTData));
	CUDA_CHECK_RETURN(hipFree(gpuINTTData));
	CUDA_CHECK_RETURN(hipFree(gpuTwid));
	delete[] in;
	delete[] NTT_out;
	delete[] INTT_out;
	return;
}

// @}


/* \name Array initializer function */
void initialize(uint32_t* in, int mod, const int size)
{
	for (int i = 0; i < size; i++)
	{
		in[i] = (uint32_t)(rand() % mod);
	}
}

// @}

int main(int argc, char *argv[])
{
	std::cerr << "NTT FFT" << std::endl;
	const int NTT_SIZE = atoi(argv[1]);	// N - number of elements to be processed
	const int BITS_PER_ELEM = atoi(argv[2]);	// n - number of bits per array element
	const int TH_LIM = atoi(argv[3]);	// maximum block size allowable per kernel
	// check if both of the required arguments are powers of two
	double chk1 = log2((double)NTT_SIZE);
	double chk2 = log2((double)BITS_PER_ELEM);
	double chk3 = (double)logf(TH_LIM) / logf(2);
	if (argc != 4 || chk1 != round(chk1) || chk2 != round(chk2) || chk3 != round(chk3) || TH_LIM < 0 || TH_LIM > 1024 || BITS_PER_ELEM > 16
			|| NTT_SIZE/TH_LIM > TH_LIM || NTT_SIZE <= 1 || BITS_PER_ELEM <= 1)
	{
	  std::cerr << "Usage: " << argv[0]
			<< " <NTT size> <Bits per element> <thread limit per block>" << std::endl;
	  exit(1);
	}
	std::cout << "Computing..." << std::endl;
	ProcessandTime(NTT_SIZE, BITS_PER_ELEM, TH_LIM);
	return 0;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}
