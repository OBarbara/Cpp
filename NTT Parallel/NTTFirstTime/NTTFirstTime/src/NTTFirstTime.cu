/*
 ============================================================================
 Name        : NTTFFTCUDA.cu
 Author      : Owen
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <stdint.h>
#include <math.h>

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
void ProcessandTime(const int size, const int bpe, const int thread_limit);
void initialize(uint32_t* arr, int size, const int bpe);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)
/**
 * CUDA kernel that computes the integer FFT of an array of unsigned integers
 */
// both of the following constants must be a power of two
// the first part takes care of the first stages before the shuffling and the
// element transfer from shared memory to global memory
__global__ void MulSMNTTKernel1(uint32_t *data, uint32_t* final_temp, uint32_t* temp_data, int curr_k, int k, int mod, int g, bool repos) {
	unsigned short idx = threadIdx.x;
	unsigned short b = blockIdx.x;
	short gridSizeSh = logf(gridDim.x) / logf(2);
	const int BLOCK_SIZE = blockDim.x;	// the offset given for the second part of shared memory
	if (idx < BLOCK_SIZE){	// for maximum performance, the last idx should be a multiple of 32 (minus 1 since it starts from 0)
		uint32_t tempvar; 	  // a temporary variable to manage subtractions and hold temporary data
		short i = idx / 2;
		short j = idx % 2;	// even or odd thread
		// used for twiddle factor indexing
		short l = k - curr_k - 1;
		int off = b * BLOCK_SIZE;
		short m = i % 2;	// helps in indexing for each thread in a block
		short b_twid = b << (l - 1);	// twiddle factor indexing for each block
		short elem = b + (idx << gridSizeSh);
		short p, sign, tmp, x, x1, twid_1, twid_2, twid_index;

		// copy data from global memory to shared memory
		final_temp[idx + off] = data[elem];
		__syncthreads();	// ensure that the shared memory is properly populated


		// stg is the stage (or epoch)
		// k is the number of stages in the computation
		for(short stg = 1; stg <= curr_k; stg++){
			p = BLOCK_SIZE >> stg;	// this variable will help in indexing
			// sign takes care of the operations for even (addition)
			// and odd (subtraction) threads
			sign = ((-2) * j) + 1;  // can be 1 or -1
			// indexing
			tmp = i + ((i >> (k - curr_k - stg)) * (1 << (k - curr_k - stg)));
			x = tmp + (j * p);
			x1 = x + (sign * p);
			twid_1 = (b + (i << gridSizeSh)) * (curr_k - stg);		// twiddle for stage 1
			twid_2 = ((l - 1) * ((!m * b_twid) + (m * (b_twid + BLOCK_SIZE))) + (curr_k - l) * (b << l)) * (stg - 1);	// twiddle for stage 2
			// twid_index manages the indexing of the twiddle factors
			twid_index =  twid_1 + twid_2;
			// since the value should be unsigned, a subtraction cannot result in a negative number
			// so we add the modulus to the number being subtracted to prevent that from happening
			tempvar = final_temp[off + x1];

			if(j != 0 && tempvar < final_temp[off + x])
				tempvar += mod;
			// addition and subtraction is taken care of here
			// modulus is done after addition/subtraction
			temp_data[off + x] = (tempvar + (sign * final_temp[off + x])) % mod;
			// shift by twiddle factor and perform modulus
			if(repos && j && twid_index)
				temp_data[off + x] <<= g * twid_index;
			final_temp[off + x] = temp_data[off + x] % mod;
			__syncthreads();
		}
		// shuffle data from shared to global memory
		data[elem] = final_temp[off + idx];
	}
}

// the first part takes care of the first stages before the shuffling and the
// element transfer from shared memory to global memory
__global__ void MulSMNTTKernel2(uint32_t *data, uint32_t* final_temp, uint32_t* temp_data, int NTT_SIZE, int curr_k, int k, int mod, int g, bool repos) {
	unsigned short idx = threadIdx.x;
	unsigned short b = blockIdx.x;
	unsigned short glob_idx = b * blockDim.x + idx;
	short gridSizeSh = logf(gridDim.x) / logf(2);
	const int BLOCK_SIZE = blockDim.x;	// the offset given for the second part of shared memory
	if (idx < BLOCK_SIZE){	// for maximum performance, the last idx should be a multiple of 32 (minus 1 since it starts from 0)
		uint32_t tempvar; 	  // a temporary variable to manage subtractions and hold temporary data
		int off = b * BLOCK_SIZE;
		short i = idx / 2;
		short j = idx % 2;	// even or odd thread
		short p, sign, tmp, x, x1, twid_index;

		// copy data
		final_temp[idx + off] = data[glob_idx];
		__syncthreads();	// ensure that the shared memory is properly populated

		// stg is the stage (or epoch)
		// k is the number of stages in the computation
		for(short stg = curr_k; stg <= k; stg++){
			p = NTT_SIZE >> stg;	// this variable will help in indexing
			// sign takes care of the operations for even (addition)
			// and odd (subtraction) threads
			sign = ((-2) * j) + 1;  // can be 1 or -1
			// indexing
			tmp = i + ((i >> (k - stg)) * (1 << (k - stg)));
			x = tmp + (j * p);
			x1 = x + (sign * p);
			// twid_index manages the indexing of the twiddle factors
			twid_index = (i % p) << (stg - 1);
			// since the value should be unsigned, a subtraction cannot result in a negative number
			// so we add the modulus to the number being subtracted to prevent that from happening
			tempvar = final_temp[off + x1];

			if(j != 0 && tempvar < final_temp[off + x])
				tempvar += mod;
			// addition and subtraction is taken care of here
			// modulus is done after addition/subtraction
			temp_data[off + x] = (tempvar + (sign * final_temp[off + x])) % mod;
			// shift by twiddle factor and perform modulus
			if(repos && j && twid_index)
				temp_data[off + x] <<= g * twid_index;
			final_temp[off + x] = temp_data[off + x] % mod;
			__syncthreads();
		}
		// write finished data from shared to global memory
		data[glob_idx] = final_temp[off + idx];
	}
}

// kernel for the inverse NTT FFT
__global__ void MulSMINTTKernel1(uint32_t *data, uint32_t* final_temp, uint32_t* temp_data, const int NTT_SIZE, const int BITS_PER_ELEM, int curr_k, int k, int mod, int g, bool repos, bool fin) {
	unsigned short idx = threadIdx.x;
	unsigned short b = blockIdx.x;
	unsigned short glob_idx = b * blockDim.x + idx;
	short gridSizeSh = logf(gridDim.x) / logf(2);
	const int BLOCK_SIZE = NTT_SIZE >> gridSizeSh;	// the offset given for the second part of shared memory
	if (idx < BLOCK_SIZE){	// for maximum performance, the last idx should be a multiple of 32 (minus 1 since it starts from 0)
		uint32_t tempvar; 	  // a temporary variable to manage subtractions and hold temporary data
		int off = b * BLOCK_SIZE;
		short i = idx / 2;
		short j = idx % 2;	// even or odd thread
		// short l = (b * blockDim.x + idx) >> 1; 	// used for finding twiddle factors
		short p, sign, tmp, x, x1, twid_index;

		// copy data
		final_temp[idx + off] = data[glob_idx];
		__syncthreads();	// ensure that the shared memory is properly populated

		// stg is the stage (or epoch)
		// k is the number of stages in the computation
		for(short stg = k; stg >= curr_k; stg--){
			p = NTT_SIZE >> stg;	// this variable will help in indexing
			// sign takes care of the operations for even (addition)
			// and odd (subtraction) threads
			sign = ((-2) * j) + 1;  // can be 1 or -1
			// indexing
			tmp = i + ((i >> (k - stg)) * (1 << (k - stg)));
			x = tmp + (j * p);
			x1 = x + (sign * p);
			// twid_index manages the indexing of the twiddle factors
			twid_index = (i % p) << (stg - 1);
			if(repos && twid_index){
				// shift by twiddle factor and perform modulus
				tempvar = (final_temp[off + x1] << (!j * g * twid_index)) % mod;
				temp_data[off + x] = final_temp[off + x] << (j * g * twid_index);
				temp_data[off + x] %= mod;
			}
			else{
				tempvar = final_temp[off + x1];
				temp_data[off + x] = final_temp[off + x];
			}

			// since the value should be unsigned, a subtraction cannot result in a negative number
			// so we add the modulus to the number being subtracted to prevent that from happening
			while(j != 0 && tempvar < temp_data[off + x])
				tempvar += mod;
			// addition and subtraction is taken care of here
			// modulus is done after addition/subtraction
			temp_data[off + x] = (tempvar + (sign * temp_data[off + x])) % mod;
			final_temp[off + x] = temp_data[off + x];
			// new data is ready for next stage
			__syncthreads();
		}

		if(fin){	// if there is only 1 SM
			// divide each element by N
			short t = (BITS_PER_ELEM << 1) - k; // convert division into multiplication
			uint32_t ls, rs;
			short size = sizeof(uint32_t) << 3; // multiply 4 bytes by 8 in this case (32 bits)
			bool rt_shift = false;	// should the shift be to the right and rotate bits?
			if(t < 0){	// if t is negative, all shifts are to the right
				uint32_t mask = t >> (size - 1);     // make a mask of the sign bit
				t ^= mask;                   // toggle the bits
				t += mask & 1;               // add one
				rt_shift = true;
			}
			short temp_sh;
			short shift = size - (BITS_PER_ELEM + 1);
			// shifting will be done in an optimized manner
			if (shift > t)
				shift = t;	// if t is less than the maximum shift amount, then assign the shift amount to be t
			temp_sh = shift;
			for (short m = 0; m < t; m += shift) {
				shift = temp_sh;	// assign the shift value from the previous iteration

				// there is no thread divergence here since all threads execute the same branch
				if(!rt_shift){	// normal left shift
					final_temp[off + idx] <<= shift;
				}
				else{	// right shift and bit rotation
					rs = final_temp[off + idx] >> shift;
					ls = final_temp[off + idx] << (size - shift);
					final_temp[off + idx] = rs | ls;
				}
				final_temp[off + idx] %= mod;
				if (t - shift <= (BITS_PER_ELEM - 1))
					temp_sh = t - shift;	// all remaining shifts will be done after modulus
			}

			// copy final data from shared memory to global memory
			if(repos)
				data[(NTT_SIZE - glob_idx) % NTT_SIZE] = final_temp[off + idx];
			else
				data[glob_idx] = final_temp[off + idx];
		}
		else{
			// copy from shared memory to global memory in a coalesced manner
			data[glob_idx] = final_temp[off + idx];
		}
	}
}

__global__ void MulSMINTTKernel2(uint32_t *data, uint32_t* final_temp, uint32_t* temp_data, const int NTT_SIZE, const int BITS_PER_ELEM, int curr_k, int k, int mod, int g, bool repos) {
	unsigned short idx = threadIdx.x;
	unsigned short b = blockIdx.x;
	short gridSizeSh = logf(gridDim.x) / logf(2);
	const int BLOCK_SIZE = blockDim.x;	// the offset given for the second part of shared memory
	if (idx < BLOCK_SIZE){	// for maximum performance, the last idx should be a multiple of 32 (minus 1 since it starts from 0)
		uint32_t tempvar; 	  // a temporary variable to manage subtractions and hold temporary data
		int off = b * BLOCK_SIZE;
		short i = idx / 2;
		short j = idx % 2;	// even or odd thread
		short l = k - curr_k - 1;
		short m = i % 2;
		short b_twid = b << (l - 1);	// twiddle factor indexing for each block
		short elem = b + (idx << gridSizeSh);
		short p, sign, tmp, x, x1, twid_1, twid_2, twid_index;

		// copy data
		final_temp[idx + off] = data[elem];
		__syncthreads();	// ensure that the shared memory is properly populated

		// stg is the stage (or epoch)
		// k is the number of stages in the computation
		for(short stg = curr_k; stg >= 1; stg--){
			p = BLOCK_SIZE >> stg;	// this variable will help in indexing
			// sign takes care of the operations for even (addition)
			// and odd (subtraction) threads
			sign = ((-2) * j) + 1;  // can be 1 or -1
			// indexing
			tmp = i + ((i >> (k - curr_k - stg)) * (1 << (k - curr_k - stg)));
			x = tmp + (j * p);
			x1 = x + (sign * p);
			twid_1 = (b + (i << gridSizeSh)) * (curr_k - stg);		// twiddle for stage 1
			twid_2 = ((l - 1) * ((!m * b_twid) + (m * (b_twid + BLOCK_SIZE))) + (curr_k - l) * (b << l)) * (stg - 1);	// twiddle for stage 2

			// twid_index manages the indexing of the twiddle factors
			twid_index =  twid_1 + twid_2;
			// shift by twiddle factor and perform modulus
			if(repos && twid_index){
				tempvar = (final_temp[off + x1] << (!j * g * twid_index)) % mod;
				temp_data[off + x] = final_temp[off + x] << (j * g * twid_index);
				temp_data[off + x] %= mod;
			}
			else{
				tempvar = final_temp[off + x1];
				temp_data[off + x] = final_temp[off + x];
			}

			// since the value should be unsigned, a subtraction cannot result in a negative number
			// so we add the modulus to the number being subtracted to prevent that from happening
			while(j != 0 && tempvar < temp_data[off + x])
					tempvar += mod;
			// addition and subtraction is taken care of here
			// modulus is done after addition/subtraction
			temp_data[off + x] = (tempvar + (sign * temp_data[off + x])) % mod;
			final_temp[off + x] = temp_data[off + x];
			// new data is ready for next stage
			__syncthreads();
		}

		// divide each element by N
		short t = (BITS_PER_ELEM << 1) - k; // convert division into multiplication
		uint32_t ls, rs;
		short size = sizeof(uint32_t) << 3; // multiply 4 bytes by 8 in this case (32 bits)
		bool rt_shift = false;	// should the shift be to the right and rotate bits?
		if(t < 0){	// if t is negative, all shifts are to the right
			uint32_t mask = t >> (size - 1);     // make a mask of the sign bit
			t ^= mask;                   // toggle the bits
			t += mask & 1;               // add one
			rt_shift = true;
		}
		short temp_sh;
		short shift = size - (BITS_PER_ELEM + 1);
		// shifting will be done in an optimized manner
		if (shift > t)
			shift = t;	// if t is less than the maximum shift amount, then assign the shift amount to be t
		temp_sh = shift;
		for (short m = 0; m < t; m += shift) {
			shift = temp_sh;	// assign the shift value from the previous iteration

			// there is no thread divergence here since all threads execute the same branch
			if(!rt_shift){	// normal left shift
				final_temp[off + idx] <<= shift;
			}
			else{	// right shift and bit rotation
				rs = final_temp[off + idx] >> shift;
				ls = final_temp[off + idx] << (size - shift);
				final_temp[off + idx] = rs | ls;
			}
			final_temp[off + idx] %= mod;
			if (t - shift <= (BITS_PER_ELEM - 1))
				temp_sh = t - shift;	// all remaining shifts will be done after modulus
		}

		// copy final data from shared memory to global memory
		if(repos)
			data[(NTT_SIZE - elem) % NTT_SIZE] = final_temp[off + idx];
		else
			data[elem] = final_temp[off + idx];
	}
}

/*! \name Timing function */

//! Returns current time in seconds
inline double gettime()
{
	struct timeval tv;
	struct timezone tz;
	gettimeofday(&tv, &tz);
	return ((double) tv.tv_sec + (double) tv.tv_usec * 1E-6);
}

// @}

/*! \name GPU function caller and process timing function */
void ProcessandTime(const int size, const int bpe, const int thlimit){
	// define NTT variables
	double intervalNTT = 0, intervalINTT = 0;
	float NTTms, iNTTms;
	// declare and create events
	hipEvent_t NTTstart, NTTstop, INTTstart, INTTstop;
	hipEventCreate(&NTTstart);
	hipEventCreate(&NTTstop);
	hipEventCreate(&INTTstart);
	hipEventCreate(&INTTstop);
	int kt = log2((double)size);
	int i;	// index for traversing through the arrays
	int modulus = (1 << bpe) + 1; // prime number for modulo arithmetic
	int rt_unity = 2 * bpe / size; // calculate # bits for root of unity
	// these flags determine whether the IFFT repositions the elements
	// and whether the result of the IFFT matches the input array or not, respectively
	bool repos_flag = true, flag = true;
	int runs = 1E3;	// number of times each kernel function is executed
	uint32_t *in = new uint32_t[size];
	uint32_t *NTT_out = new uint32_t[size];
	uint32_t *INTT_out = new uint32_t[size];
	uint32_t* gpuNTTData, *gpuINTTData, *gpuTwid, *gputempData1, *gputempData2;
	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuNTTData, sizeof(uint32_t) * size));
	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuTwid, sizeof(uint32_t)*size / 2));
	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuINTTData, sizeof(uint32_t)*size));
	CUDA_CHECK_RETURN(hipMalloc((void **)&gputempData1, sizeof(uint32_t)*size));
	CUDA_CHECK_RETURN(hipMalloc((void **)&gputempData2, sizeof(uint32_t)*size));
	uint32_t twiddle[size / 2];	// pre-compute twiddle factor array
	if(!rt_unity)
		repos_flag = false;

	int blk_size;
	bool isSingleSM = false;
	if(size <= thlimit){
		blk_size = size;
		isSingleSM = true;
	}
	else
		blk_size = thlimit;
	static const int BLOCK_SIZE = blk_size;	// amount of threads in each block
	const int blockCount = (size) / BLOCK_SIZE;	// amount of blocks in a grid
	short k = log2((double)blockCount);	// first kernel covers first k stages only out of kt for FFT, and vice-versa for IFFT kernels
	std::cout << "Launching kernels with " << blockCount << " block(s), each with " << BLOCK_SIZE << " threads." << std::endl;
	srand(time(NULL));	// generate the seed for the pseudo-random number generator
	for (int j = 0; j < runs; j++) {
		initialize(in, modulus, size);

		CUDA_CHECK_RETURN(hipMemcpy(gpuNTTData, in, sizeof(uint32_t)*size, hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMemcpy(gpuTwid, twiddle, sizeof(uint32_t)*size/2, hipMemcpyHostToDevice));

		hipEventRecord(NTTstart);
		if(!isSingleSM){
			// computes first k stages and shuffles data from shared memory to global memory
			MulSMNTTKernel1<<<blockCount, BLOCK_SIZE, BLOCK_SIZE * sizeof(uint32_t)>>> (gpuNTTData, gputempData1, gputempData2, k, kt, modulus, rt_unity, repos_flag);
		}
		// shuffles data from global memory to shared memory and computes last set of stages
		MulSMNTTKernel2<<<blockCount, BLOCK_SIZE, BLOCK_SIZE * sizeof(uint32_t)>>> (gpuNTTData, gputempData1, gputempData2, size, k + 1, kt, modulus, rt_unity, repos_flag);
		hipEventRecord(NTTstop);
		hipEventSynchronize(NTTstop);
		hipEventElapsedTime(&NTTms, NTTstart, NTTstop);

		CUDA_CHECK_RETURN(hipMemcpy(NTT_out, gpuNTTData, sizeof(uint32_t) * size, hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpy(gpuINTTData, NTT_out, sizeof(uint32_t)*size, hipMemcpyHostToDevice));

		hipEventRecord(INTTstart);
		// computes first stages and shuffles data from shared memory to global memory
		MulSMINTTKernel1<<<blockCount, BLOCK_SIZE, BLOCK_SIZE * sizeof(uint32_t)>>> (gpuINTTData, gputempData1, gputempData2, size, bpe, k + 1, kt, modulus, rt_unity, repos_flag, isSingleSM);
		if(!isSingleSM){
			// shuffles data from global memory to shared memory and computes last set of stages
			MulSMINTTKernel2<<<blockCount, BLOCK_SIZE, BLOCK_SIZE * sizeof(uint32_t)>>> (gpuINTTData, gputempData1, gputempData2, size, bpe, k, kt, modulus, rt_unity, repos_flag);
		}
		hipEventRecord(INTTstop);

		CUDA_CHECK_RETURN(hipMemcpy(INTT_out, gpuINTTData, sizeof(uint32_t)*size, hipMemcpyDeviceToHost));

		hipEventSynchronize(INTTstop);
		hipEventElapsedTime(&iNTTms, INTTstart, INTTstop);

		// total time taken is summed up for all runs
		intervalNTT += NTTms;
		intervalINTT += iNTTms;


		for (i = 0; i < size; i++) {
			if (in[i] != INTT_out[i]) {
				flag = false;
				break;
			}
		}

		if (!flag) {
			std::cout << "j = " << j << ", i = " << i << "\nin = " << in[i] << ", out = " << INTT_out[i] << std::endl;
			break;
		}
	}

	if (flag){
		std::cout << "NTT matched. Average time taken for NTT is: " << intervalNTT << " seconds" << std::endl;
		std::cout << "Average time taken for INTT is: " << intervalINTT << " seconds" << std::endl;
	}

	// clean up
	CUDA_CHECK_RETURN(hipFree(gpuNTTData));
	CUDA_CHECK_RETURN(hipFree(gpuINTTData));
	CUDA_CHECK_RETURN(hipFree(gputempData1));
	CUDA_CHECK_RETURN(hipFree(gputempData2));
	CUDA_CHECK_RETURN(hipFree(gpuTwid));
	delete[] in;
	delete[] NTT_out;
	delete[] INTT_out;
	return;
}

// @}


/* \name Array initializer function */
void initialize(uint32_t* in, int mod, const int size)
{
	for (int i = 0; i < size; i++)
	{
		in[i] = (uint32_t)(rand() % mod);
	}
}

// @}

int main(int argc, char *argv[])
{
	std::cerr << "NTT FFT" << std::endl;
	const int NTT_SIZE = atoi(argv[1]); // number of elements to be processed (N)
	const int BITS_PER_ELEM = atoi(argv[2]); // number of bits to represent one element (n)
	const int TH_LIM = atoi(argv[3]); // maximum number of threads allowable per kernel
	// check if both of the required arguments are powers of two
	double chk1 = log2((double)NTT_SIZE);
	double chk2 = log2((double)BITS_PER_ELEM);
	double chk3 = (double)logf(TH_LIM) / logf(2);
	// check whether all user parameters are valid
	if (argc != 4 || chk1 != round(chk1) || chk2 != round(chk2) || chk3 != round(chk3) || TH_LIM < 0 || TH_LIM > 1024 || BITS_PER_ELEM > 16
				|| NTT_SIZE <= 1 || BITS_PER_ELEM <= 1)
	{
	  std::cerr << "Usage: " << argv[0]
			<< " <NTT size> <Bits per element> <thread limit per block>" << std::endl;
	  exit(1);
	}
	std::cout << "Computing..." << std::endl;
	ProcessandTime(NTT_SIZE, BITS_PER_ELEM, TH_LIM);
	return 0;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}
